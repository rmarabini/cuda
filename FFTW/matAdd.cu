#include "hip/hip_runtime.h"
/* File:     mat_add.cu
 * Purpose:  Implement matrix addition on a gpu using cuda
 *
 * Output:   Result of matrix addition.  
 *
 * Notes:
 * 1.  There are m blocks with n threads each.
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"
 //#include "utils/cuPrintf.cu"
//#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
//#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

#include "fftw3.h"


/*---------------------------------------------------------------------
 * Kernel:   Mat_add
 * Purpose:  Implement matrix addition
 * In args:  A, B, m, n
 * Out arg:  C
 */

__global__ void fftwFunc(float matIn[], 
                           float matOut[], 
                           int dimX, 
                           int dimY, 
                           float rotMat[]) {
//    int y = blockIdx.y;
///    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if ( x >= dimX || y >= dimY) 
         return;

    //// compute target address
    int  x0=dimX/2, y0=dimY/2;
    const unsigned int idx = x + y * dimX;

    const int xA = (x - x0 );
    const int yA = (y - y0 );

    const float xR = ( xA * rotMat[0] - yA * rotMat[1]);
    const float yR = ( -xA * rotMat[2] + yA * rotMat[3]);
    float src_x = xR + x0;
    float src_y = yR + y0;

     if ( src_x >= 0.0f && src_x < dimX && src_y >= 0.0f && src_y < dimY) {
        // BI - LINEAR INTERPOLATION
        float src_x0 = (float)(int)(src_x);
        float src_x1 = (src_x0+1); if(src_x1 == dimX) src_x1=src_x0;
        float src_y0 = (float)(int)(src_y);
        float src_y1 = (src_y0+1); if(src_y1 == dimY) src_y1=src_y0;

        float sx = (src_x-src_x0);
        float sy = (src_y-src_y0);


        int idx_src00 = min(src_x0   + src_y0 * dimX,dimX*dimY-1.0f);
        int idx_src10 = min(src_x1   + src_y0 * dimX,dimX*dimY-1.0f);
        int idx_src01 = min(src_x0   + src_y1 * dimX,dimX*dimY-1.0f);
        int idx_src11 = min(src_x1   + src_y1 * dimX,dimX*dimY-1.0f);

        matOut[idx]  = (1.0f-sx)*(1.0f-sy)*matIn[idx_src00];
        matOut[idx] += (     sx)*(1.0f-sy)*matIn[idx_src10];
        matOut[idx] += (1.0f-sx)*(     sy)*matIn[idx_src01];
        matOut[idx] += (     sx)*(     sy)*matIn[idx_src11];
    } else {
        matOut[idx] = 0.0f;
     }




}  /* Mat_add */


/*---------------------------------------------------------------------
 * Function:  Fill_matrix
 * Purpose:   Fill an m x n matrix with random values
 * In args:   m, n
 * Out arg:   A
 */
void Fill_matrix(float A[], int dimX, int dimY) {
   int i, j;
//numVec, dimVec
   for (i = 0; i < dimX; i++)
      for (j = 0; j < dimY; j++)
          if(i==j )//or (i+j)==(dimX+1))
            A[i*dimY+j]=1.0f;
          else
            A[i*dimY+j]=0.0f;
}  /* Read_matrix */


/*---------------------------------------------------------------------
 * Function:  Print_matrix
 * Purpose:   Print an m x n matrix to stdout
 * In args:   title, A, m, n
 */
void Print_matrix(const char title[], float A[], int numVec, int dimVec, int m, int n) {
   int i, j;
   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.1f ", A[i*dimVec+j]);
      printf("\n");
   }  
}  /* Print_matrix */

void Print_matrix_complex(const char title[], fftwf_complex A[], int numVec, int dimVec, int m, int n) {
   int i, j;
   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.2f-i%.2f ", A[i*dimVec+j][0],A[i*dimVec+j][1] );
      printf("\n");
   }  
}  /* Print_matrix */

void checkError(hipError_t error, const char function[])
{

        if(error != hipSuccess)
        {
                printf("\"%s\" has a problem with error code %d and desc: %s\n", function, error, hipGetErrorString(error));
                exit(-1);
        }
}

bool checkIfMatricesEqual(fftwf_complex * mat1, fftwf_complex * mat2, float matSize)
{
    int i = 0;
    for( ; i < matSize; i++)
       if(mat1[i] != mat2[i]){
           printf("values different for i: %d\n", i);
		   printf("mat1[i] = %d, mat2[i] = %d\n", mat1[i], mat2[i]);		   
		   return false;
	   }

    return true;
}
void fftwCPU(float matIn[], 
               fftwf_complex matOut[], int dimX, int dimY){
//double * image=(double *) malloc(640*480*sizeof(double));
//fftw_complex * out2d=(fftw_complex *)
//malloc(640*480*sizeof(fftw_complex));
//fftw_complex * out2c=(fftw_complex *)
//malloc(640*480*sizeof(fftw_complex));
  
    fftwf_plan p2d;
    int n[2];
    n[0]=dimX; n[1]=dimY;
    //p2d = fftwf_plan_dft_r2c_2d(dimX, dimY, matIn,matOut,FFTW_ESTIMATE );
    p2d = fftwf_plan_dft_r2c(2, n, matIn,matOut,FFTW_ESTIMATE );
    fftwf_execute(p2d);
}

/* Host code */
int main(int argc, char* argv[]) {
   size_t dimX = 3;//mat size
   size_t dimY = 3;

   // variables for threads per block, number of blocks.
   int threadsPerBlockX = 32;//, blocksInGrid = 0;   
   int threadsPerBlockY = 32;//, blocksInGrid = 0;

   //threadsPerBlock = min(_dimY, _dimY);
   //create cuda event variables
   hipEvent_t hostStart, hostStop, deviceStart, deviceStop;
   float timeDifferenceOnHost, timeDifferenceOnDevice;
   //initialize cuda timing variables
   hipEventCreate(&hostStart);
   hipEventCreate(&hostStop);
   hipEventCreate(&deviceStart);
   hipEventCreate(&deviceStop);

   float *h_A;//PC
   float *d_A, *d_B;//GPU
   size_t size, matrixSize;

   /* Get size of matrices */

   matrixSize = dimX*dimY;
   size = matrixSize*sizeof(float);

   h_A = (float*) calloc(size,1);
   fftwf_complex * h_B=(fftwf_complex *) malloc(size);
   fftwf_complex * h_B2=(fftwf_complex *) malloc(size);

   /* Allocate matrices in device memory */
   hipMalloc(&d_A, size);
   hipMalloc(&d_B, size*2);

   Fill_matrix(h_A, dimX, dimY);
   Print_matrix("original matrix is: ", h_A, dimX, dimY, 3, 3);
      printf("fftw on CPU...\n");
      hipEventRecord(hostStart, 0);
      //rotate matrix using CPU
      //memset(h_B2, 0, size);
      fftwCPU(h_A ,h_B2, dimX, dimY);
      Print_matrix_complex("The fft image(CPU) is: ", h_B2, dimX, dimY, 3, 3);
      return;      
      hipEventRecord(hostStop, 0);
      hipEventElapsedTime(&timeDifferenceOnHost, hostStart, hostStop);
      printf("Matrix fft over. Time taken on CPU: %5.5f\n",     
          timeDifferenceOnHost);

      /* Copy matrices from host memory to device memory */
      memset(h_B, 0, size);
      hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
      hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


      /* Invoke kernel using dimX * dimY thread blocks, each of    */
      /* which contains threadsPerBlock threads                        */
      dim3 block(threadsPerBlockX, threadsPerBlockY);   
      dim3 grid;
      grid.x = (dimX + block.x - 1)/block.x;
      grid.y = (dimY + block.y - 1)/block.y;
      hipEventRecord(deviceStart, 0);
      //////////////fftwFunc<<<grid, block>>>(d_A, d_B, dimX, dimY, d_rotMat);
      hipError_t code=hipGetLastError();
      if (code)
         printf("error=%s",hipGetErrorString(code));
      hipDeviceSynchronize();  
      hipEventRecord(deviceStop, 0);

      /* Wait for the kernel to complete */
      hipDeviceSynchronize();
      hipEventElapsedTime(&timeDifferenceOnDevice, deviceStart, deviceStop);

      /* Copy result from device memory to host memory */
      checkError(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost), "Matrix B Copy from device to Host");
      if(checkIfMatricesEqual(h_B, h_B2, matrixSize))
          printf("Kernels correct!\n");
      else
         printf("Kernel logic wrong!\n");
	
      printf("Finished fft on GPU. Time taken: %5.5f\n", timeDifferenceOnDevice);   
      printf("Speedup: %5.5f\n", (float)timeDifferenceOnHost/timeDifferenceOnDevice);
      printf("GPUtime: %5.5f\n", (float)timeDifferenceOnDevice);

      Print_matrix_complex("The fft image(CPU) is: ", h_B2, dimX, dimY, 3, 3);
      //Print_matrix("The fft image(GPU) is: ", h_B, dimX, dimY, 9, 9);
      
   /* Free device memory */
   hipFree(d_A);
   hipFree(d_B);

   /* Free host memory */
   free(h_A);
   free(h_B);
   free(h_B2);

   return 0;
}  /* main */
