#include "hip/hip_runtime.h"
/* File:     mat_add.cu
 * Purpose:  Implement matrix addition on a gpu using cuda
 *
 * Output:   Result of matrix addition.  
 *
 * Notes:
 * 1.  There are m blocks with n threads each.
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
//#include "cuPrintf.cuh"
//#include "cuPrintf.cu"
 //#include "utils/cuPrintf.cu"
//#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
//#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

#include "fftw3.h"
#include <complex.h>
#include <hipfft/hipfft.h>


/*---------------------------------------------------------------------
 * Kernel:   Mat_add
 * Purpose:  Implement matrix addition
 * In args:  A, B, m, n
 * Out arg:  C
 */


/*---------------------------------------------------------------------
 * Function:  Fill_matrix
 * Purpose:   Fill an m x n matrix with random values
 * In args:   m, n
 * Out arg:   A
 */
void Fill_matrix(float A[], int dimX, int dimY) {
   int i, j;
//numVec, dimVec
   for (i = 0; i < dimX; i++)
      for (j = 0; j < dimY; j++)
          if(i==j )//or (i+j)==(dimX+1))
            A[i*dimY+j]=1.0f;
          else
            A[i*dimY+j]=0.0f;
}  /* Read_matrix */


/*---------------------------------------------------------------------
 * Function:  Print_matrix
 * Purpose:   Print an m x n matrix to stdout
 * In args:   title, A, m, n
 */
void Print_matrix(const char title[], float A[], int numVec, int dimVec, int m, int n) {
   int i, j;
   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.1f ", A[i*dimVec+j]);
      printf("\n");
   }  
}  /* Print_matrix */

void Print_matrix_complex(const char title[], fftwf_complex A[], int dimY, int dimX, int m, int n) {
   int i, j;
   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++){
           //c=A[i*dimX+j];
         printf("%.2f%+.2fi ", A[i*dimX+j][0], A[i*dimX+j][1]);
               }
      printf("\n");
   }  
}  /* Print_matrix */

void checkError(hipError_t error, const char function[])
{

        if(error != hipSuccess)
        {
                printf("\"%s\" has a problem with error code %d and desc: %s\n", function, error, hipGetErrorString(error));
                exit(-1);
        }
}

bool checkIfMatricesEqual(fftwf_complex * mat1, fftwf_complex * mat2, float matSize)
{
    int i = 0;
    for( ; i < matSize; i++)
       if(mat1[i] != mat2[i]){
           printf("values different for i: %d\n", i);
		   printf("mat1[i] = %d, mat2[i] = %d\n", mat1[i], mat2[i]);		   
		   return false;
	   }

    return true;
}
void fftwCPU(float matIn[], 
               fftwf_complex matOut[], int dimX, int dimY){  
    fftwf_plan p2d;
    p2d = fftwf_plan_dft_r2c_2d(dimX, dimY, matIn,matOut,FFTW_ESTIMATE );
    fftwf_execute(p2d);
}

/* Host code */
int main(int argc, char* argv[]) {
   size_t dimX = 3;//mat size
   size_t dimY = 3;

   // variables for threads per block, number of blocks.
   int threadsPerBlockX = 32;//, blocksInGrid = 0;   
   int threadsPerBlockY = 32;//, blocksInGrid = 0;

   //threadsPerBlock = min(_dimY, _dimY);
   //create cuda event variables
   hipEvent_t hostStart, hostStop, deviceStart, deviceStop;
   float timeDifferenceOnHost, timeDifferenceOnDevice;
   //initialize cuda timing variables
   hipEventCreate(&hostStart);
   hipEventCreate(&hostStop);
   hipEventCreate(&deviceStart);
   hipEventCreate(&deviceStop);

   float *h_A;//PC
   float *d_A, *d_B;//GPU
   size_t size, matrixSize;

   /* Get size of matrices */

   matrixSize = dimX*dimY;
   size = matrixSize*sizeof(float);
   int sizeFourier = dimY*(dimX/2+1)*sizeof(fftwf_complex);
   //typedef float hipfftReal; is a single-precision, floating-point real data type. 
   h_A = (float*) calloc(size,1);
   hipfftComplex  * h_B  =(hipfftComplex *) malloc(sizeFourier);
   fftwf_complex * h_B2 =(fftwf_complex *) malloc(sizeFourier);

   /* Allocate matrices in device memory */
   hipMalloc(&d_A, size);
   hipMalloc(&d_B, sizeFourier);

   Fill_matrix(h_A, dimX, dimY);
   Print_matrix("original matrix is: ", h_A, dimX, dimY, 3, 3);
      printf("fftw on CPU...\n");
      hipEventRecord(hostStart, 0);
      //rotate matrix using CPU
      //memset(h_B2, 0, size);
      fftwCPU(h_A ,h_B, dimX, dimY);
      Print_matrix_complex("The fft image(CPU) is: ", h_B2, dimY, dimX/2+1, 3, 2);

      hipEventRecord(hostStop, 0);
      hipEventElapsedTime(&timeDifferenceOnHost, hostStart, hostStop);
      printf("Matrix fft over. Time taken on CPU: %5.5f\n",     
          timeDifferenceOnHost);

      //Create Plan
      hipfftHandle plan;
      hipfftPlan2d(&plan, dimX, dimY, HIPFFT_R2C);

      /* Copy matrices from host memory to device memory */
//      memset(h_B, 0, size);
      hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
      hipMemcpy(d_B, h_B, sizeFourier, hipMemcpyHostToDevice);

      hipfftExecR2C(plan, d_A, d_B);
      hipError_t code=hipGetLastError();
      if (code)
         printf("error=%s",hipGetErrorString(code));
      hipDeviceSynchronize();  
      hipEventRecord(deviceStop, 0);

      /* Wait for the kernel to complete */
      hipDeviceSynchronize();
      hipEventElapsedTime(&timeDifferenceOnDevice, deviceStart, deviceStop);

      /* Copy result from device memory to host memory */
      checkError(hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost), "Matrix B Copy from device to Host");
/*
      if(checkIfMatricesEqual(h_B, h_B2, matrixSize))
          printf("Kernels correct!\n");
      else
         printf("Kernel logic wrong!\n");
*/	
      printf("Finished fft on GPU. Time taken: %5.5f\n", timeDifferenceOnDevice);   
      printf("Speedup: %5.5f\n", (float)timeDifferenceOnHost/timeDifferenceOnDevice);
      printf("GPUtime: %5.5f\n", (float)timeDifferenceOnDevice);

      Print_matrix_complex("The fft image(CPU) is: ", h_B2, dimX, dimY, 3, 3);
      Print_matrix("The fft image(GPU) is: ", h_B, dimX, dimY, 3, 3);
      
   /* Free device memory */
   hipFree(d_A);
   hipFree(d_B);

   /* Free host memory */
   free(h_A);
   free(h_B);
   free(h_B2);

   return 0;
}  /* main */
