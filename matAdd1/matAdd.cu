#include "hip/hip_runtime.h"
/* File:     mat_add.cu
 * Purpose:  Implement matrix addition on a gpu using cuda
 *
 * Output:   Result of matrix addition.  
 *
 * Notes:
 * 1.  There are m blocks with n threads each.
 */
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

/*---------------------------------------------------------------------
 * Kernel:   Mat_add
 * Purpose:  Implement matrix addition
 * In args:  A, B, m, n
 * Out arg:  C
 */
__global__ void Mat_add(float A[], float B[], float C[], int m, int n) {
   /* blockDim.x = threads_per_block                            */
   /* First block gets first threads_per_block components.      */
   /* Second block gets next threads_per_block components, etc. */
   int my_ij = blockDim.x * blockIdx.x + threadIdx.x;

   /* The test shouldn't be necessary */
   if (blockIdx.x < m && threadIdx.x < n) 
      C[my_ij] = A[my_ij] + B[my_ij];
}  /* Mat_add */


/*---------------------------------------------------------------------
 * Function:  Fill_matrix
 * Purpose:   Fill an m x n matrix with random values
 * In args:   m, n
 * Out arg:   A
 */
void Fill_matrix(float A[], int m, int n) {
   int i, j;

   for (i = 0; i < m; i++)
      for (j = 0; j < n; j++)
         A[i*n+j]=rand()/(float)RAND_MAX;
}  /* Read_matrix */


/*---------------------------------------------------------------------
 * Function:  Print_matrix
 * Purpose:   Print an m x n matrix to stdout
 * In args:   title, A, m, n
 */
void Print_matrix(const char title[], float A[], int m, int n) {
   int i, j;
   
   printf("%s\n", title);
   for (i = 0; i < m; i++) {
      for (j = 0; j < n; j++)
         printf("%.1f ", A[i*n+j]);
      printf("\n");
   }  
}  /* Print_matrix */

void checkError(hipError_t error, const char function[])
{

        if(error != hipSuccess)
        {
                printf("\"%s\" has a problem with error code %d and desc: %s\n", function, error, hipGetErrorString(error));
                exit(-1);
        }
}

bool checkIfMatricesEqual(float * mat1, float * mat2, float matSize)
{
    int i = 0;
    for( ; i < matSize; i++)
       if(mat1[i] != mat2[i]){
           printf("values different for i: %d\n", i);
		   printf("mat1[i] = %d, mat2[i] = %d\n", mat1[i], mat2[i]);		   
		   return false;
	   }

    return true;
}

/* Host code */
int main(int argc, char* argv[]) {
   size_t m = 1000;//mat size
   size_t n = 1000;

   // variables for threads per block, number of blocks.
   int threadsPerBlock = 16;//, blocksInGrid = 0;

   //create cuda event variables
   hipEvent_t hostStart, hostStop, deviceStart, deviceStop;
   float timeDifferenceOnHost, timeDifferenceOnDevice;
   //initialize cuda timing variables
   hipEventCreate(&hostStart);
   hipEventCreate(&hostStop);
   hipEventCreate(&deviceStart);
   hipEventCreate(&deviceStop);

   float *h_A, *h_B, *h_C, *h_C2;//PC
   float *d_A, *d_B, *d_C;//GPU
   size_t size, matrixSize;

   /* Get size of matrices */
   printf("m = %d, n = %d\n", m, n);
   matrixSize = m*n;
   size = matrixSize*sizeof(float);

   h_A = (float*) malloc(size);
   h_B = (float*) malloc(size);
   h_C = (float*) malloc(size);
   h_C2 = (float*) malloc(size);
   
   Fill_matrix(h_A, m, n);
   Fill_matrix(h_B, m, n);

   Print_matrix("A =", h_A, 4, 5);
   Print_matrix("B =", h_B, 4, 5);

   printf("Adding matrices on CPU...\n");
   hipEventRecord(hostStart, 0);
   for(int i = 0 ; i < m*n; i++)
           h_C2[i] = h_A[i] + h_B[i];
   Print_matrix("CPU =", h_C2, 4, 5);
   hipEventRecord(hostStop, 0);
   hipEventElapsedTime(&timeDifferenceOnHost, hostStart, hostStop);
   printf("Matrix addition over. Time taken on CPU: %5.5f\n",     
          timeDifferenceOnHost);


   /* Allocate matrices in device memory */
   hipMalloc(&d_A, size);
   hipMalloc(&d_B, size);
   hipMalloc(&d_C, size);

   /* Copy matrices from host memory to device memory */
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

   //create a proper grid block using dim3

   /* Invoke kernel using m thread blocks, each of    */
   /* which contains n threads                        */
   dim3 block(threadsPerBlock,threadsPerBlock);
   dim3 grid( (n + threadsPerBlock - 1/block.x), 
              (m + block.y - 1/block.y));

   hipEventRecord(deviceStart, 0);
   Mat_add<<<block, grid>>>(d_A, d_B, d_C, m, n);
   hipEventRecord(deviceStop, 0);

   /* Wait for the kernel to complete */
   hipDeviceSynchronize();
   hipEventElapsedTime(&timeDifferenceOnDevice, deviceStart, deviceStop);

   /* Copy result from device memory to host memory */
   checkError(hipMemcpy(d_C, h_C, size, hipMemcpyDeviceToHost), "Matrix C Copy from device to Host");
	
   if(checkIfMatricesEqual(h_C, h_C2, matrixSize))
      printf("Kernels correct!\n");
   else
      printf("Kernel logic wrong!\n");
	
   printf("Finished addition on GPU. Time taken: %5.5f\n", timeDifferenceOnDevice);   
   printf("Speedup: %5.5f\n", (float)timeDifferenceOnHost/timeDifferenceOnDevice);

   Print_matrix("The sum (CPU) is: ", h_C2, 4, 5);
   Print_matrix("The sum (GPU) is: ", h_C, 4, 5);

   /* Free device memory */
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   /* Free host memory */
   free(h_A);
   free(h_B);
   free(h_C);

   return 0;
}  /* main */
