#include "hip/hip_runtime.h"
/**
 * vecAdd: C = A + B.
 *  
 * Partially based on CUDA samples from CUDA 7.5 Toolkit 
 *
 */

#include <stdio.h>
#include <time.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 *
 * Error checking is not performed for simplicity
 * 
*/
__global__ void
vecAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    hipError_t error;

    // Use device 0 or 1    
    hipSetDevice(0);

    int numElements = 1000000;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host vectors A, B and C
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host input vectors
    time_t t;
    srand((unsigned) time(&t));
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device vectors A, B and C
    hipEvent_t startMem, stopMem; 

    hipEventCreate(&startMem);
    hipEventCreate(&stopMem);
    hipEventRecord(startMem, 0) ;

    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Create start and stop CUDA events to measure time
    hipEvent_t start, stop;
    float time;

    // Copy the host input vectors A and B to the device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(stopMem, 0) ;
    hipEventElapsedTime(&time, startMem, stopMem) 
    printf("time=%f,  Host to Device bandwidth (GB/s): %fn", time, numElements * 1e-6 / time);
    
    // Launch the Vector Add CUDA Kernel
///*
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );
    
      int threadsPerBlock = 1024;
      int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
      printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
      vecAdd<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_B, d_C, numElements);
      error=hipGetLastError();
      if (error!=hipSuccess)
         printf("Maaaaal!!\n!");
//    vecAdd<<<1,numElements>>>(d_A, d_B, d_C, numElements);
    
    hipEventRecord( stop, 0);
    hipEventSynchronize( stop );
    hipEventElapsedTime( &time, start, stop );

    // Copy the device result vector to the host memory.
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print kernel execution time
    printf("Kernel execution time %f\n", time);

    // Testing some values
    for (int i=0; i<10; i++)
	printf("%d -> A+B(host)=%f, A+B(GPU)=%f\n",i, h_A[i]+h_B[i], h_C[i]); 
    
    for (int i=1024; i<1034; i++)
	printf("%d -> A+B(host)=%f, A+B(GPU)=%f\n",i, h_A[i]+h_B[i], h_C[i]); 
    
    // Free device global memory (no error checking)
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Destroy event
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Reset the device and exit
    hipDeviceReset();

    printf("Done\n");
    return 0;
}

